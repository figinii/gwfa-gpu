#ifndef CHAR_GENETIC_STRUCTURE_H
#define CHAR_GENETIC_STRUCTURE_H

#include "geneticStructures.h"
#include "lcp.h"

class GeneticStrChar : public GeneticStr<GeneticStrChar> {
private:
public:
  char* content;
  u_long len;


  GeneticStrChar(){
    content = (char*) malloc(sizeof(char));
    *content = (char) "\0";
    len = 0;
  }

  GeneticStrChar(seqan3::dna5_vector sequence){
    content = (char*) malloc(sizeof(char) * (sequence.size()+1));
    int i = 0;
    for(auto nucleotide : sequence){
      char tmp = nucleotide.to_char();
      content[i] = tmp;
      i++;
    }
    len = sequence.size();
  }

  GeneticStrChar(std::string sequence){
    content = (char*) malloc(sizeof(char) * (sequence.size()+1));
    int i = 0;
    for(auto nucleotide : sequence){
      content[i] = nucleotide;
      i++;
    }
    len = sequence.size();
  }

  void GeneticStrDeviceMove(GeneticStrChar** ptr){
    char* contentD;
    hipMalloc((void**)&contentD, (sizeof(char) * (len+1)));
    hipMemcpy(contentD, this->content, (this->len+1), hipMemcpyHostToDevice);

    hipMalloc((void**)ptr, sizeof(GeneticStrChar));
    hipMemcpy(*ptr, this, sizeof(GeneticStrChar), hipMemcpyHostToDevice);
  }

  long lcp(long myIndex, GeneticStrChar* other, long otherIndex) const override {
    long len = 0;
    while(content[myIndex+len] != '\0' && 
          other->content[len+otherIndex] != '\0' &&
          content[myIndex+len] == other->content[len+otherIndex])
      len++;
    return len;
  }

  std::string toString() const override{
    return content;
  }

  ~GeneticStrChar() override{
    free(content);
  }

};

#endif