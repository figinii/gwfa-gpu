#ifndef CHAR_GENETIC_STRUCTURE_H
#define CHAR_GENETIC_STRUCTURE_H

#include "geneticStructures.h"
#include "lcp.h"
#include <hip/hip_runtime.h>
#include <iostream>

class GeneticStrChar : public GeneticStr<GeneticStrChar> {
private:
public:
  char* content;
  u_long len;


  GeneticStrChar(){
    content = (char*) malloc(sizeof(char));
    *content = '\0';
    len = 0;
  }

  GeneticStrChar(std::string sequence){
    content = (char*) malloc(sizeof(char) * (sequence.size()+1));
    int i = 0;
    for(auto nucleotide : sequence){
      content[i] = nucleotide;
      i++;
    }
    len = sequence.size();
    this->content[len] = '\0';
  }

  void sendToCuda(GeneticStrChar** ptr) const override{
    char* contentD;
    hipMalloc((void**)&contentD, (sizeof(char) * (len+1)));
    hipMemcpy(contentD, this->content, (sizeof(char) * (len+1)), hipMemcpyHostToDevice);

    hipMalloc((void**)ptr, sizeof(GeneticStrChar));
    GeneticStrChar* tmp = new GeneticStrChar();
    tmp->content = contentD;
    tmp->len = len;
    hipMemcpy(*ptr, tmp, sizeof(GeneticStrChar), hipMemcpyHostToDevice);
  }

  long lcp(long myIndex, GeneticStrChar* other, long otherIndex) const override {
    long len = 0;
    while(content[myIndex+len] != '\0' && 
          other->content[len+otherIndex] != '\0' &&
          content[myIndex+len] == other->content[len+otherIndex])
      len++;
    return len;
  }

  std::string toString() const override{
    return content;
  }

  ~GeneticStrChar() override{
    free(content);
  }

};

#endif