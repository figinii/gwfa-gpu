#include "parsingFunctions.h"
#include "charBasedGeneticStructures.cu"
#include "gwfa.hu"
#include <hip/hip_runtime.h>

#include <iostream>
#include <cstring>

int main(int argc, char **argv)
{
  hipSetDevice(1);
  if(argc < 3){
    std::cout << "Usage: " << argv[0] << "<sequenceFilePath.fa> " << " " 
                                                 << "<graphFilePath.fa> "   << std::endl;
    return 1;
  }

  // integeArray representation usage
  GeneticStrChar* sequence = new GeneticStrChar(parseFA(argv[1])[">read1"]);
  GeneticNode* first = parseGFA(argv[2]);
  TrackedGeneticNode* tracked = new TrackedGeneticNode(first, 0, 0);

  GeneticStrChar* queryD;
  TrackedGeneticNode* trackedD;
  GeneticNode* pttr;
  sequence->sendToCuda(&queryD);
  first->sendToCuda(&pttr);
  tracked->sendToCuda(&trackedD, &pttr);
  test<<<1, 1>>>(queryD);
  test<<<1, 1>>>(tracked, pttr);
  hipDeviceSynchronize();

  std::cout << tracked->col << std::endl;
  std::cout << first->nodeContent->lcp(0, sequence, 0) << std::endl;

  // std::cout << sequence->toString() << std::endl;
  // std::cout << first->nodeContent->toString() <<std::endl;
  // std::cout << first->reachableNodes[0]->nodeContent->toString() << std::endl;
  
  return 0;
}